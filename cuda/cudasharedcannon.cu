#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include "simplesharedkernel.h"

void swap(unsigned char **d_data,unsigned char **d_results)
{
	unsigned char* temp = *d_data;
	*d_data = *d_results;
	*d_results = temp;
}

int main()
{
	int size,i,imageW,imageH;
	unsigned char *h_data;
	unsigned char *h_results;

	unsigned char *d_data;
	unsigned char *d_results;

	unsigned char h_filter[9];
	h_filter[0] = 1;
	h_filter[1] = 2;
	h_filter[2] = 1;
	h_filter[3] = 2;
	h_filter[4] = 4;
	h_filter[5] = 2;
	h_filter[6] = 1;
	h_filter[7] = 2;
	h_filter[8] = 1;

	imageW = 1920;
	imageH = 2520;
	size = imageW* imageH;

	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	h_data =(unsigned char*)malloc(size);
	h_results =(unsigned char*)malloc(size);

	FILE* inputImage;
	inputImage = fopen("../image.raw","rb");
	fread(h_data,size,1,inputImage);
	fclose(inputImage);


	dim3 BLOCKDIM(BLOCKDIM_X , BLOCKDIM_Y);
	int numBlocks_X = imageW / BLOCKDIM_X;
	int numBlocks_Y = imageH / BLOCKDIM_Y;


	dim3 gridSize(numBlocks_X, numBlocks_Y);

	printf("blocks x %d blocks y %d\n",gridSize.x,gridSize.y );
	printf("blocks x %d blocks y %d\n",BLOCKDIM.x,BLOCKDIM.y );


	hipEventRecord(start, 0);

	hipMalloc(&d_data, size);
	hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice);
	hipMalloc(&d_results, size);
	setFilter(h_filter);


	for(i = 0; i < 100; i++)
	{
		filter<<<gridSize,BLOCKDIM>>>(d_data,d_results,imageW,imageH);
		swap(&d_data,&d_results);
	}

	hipMemcpy(h_results, d_results, size, hipMemcpyDeviceToHost);
	hipFree(d_results);
	hipFree(d_data);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	FILE* outputImage;
	outputImage = fopen("out.raw","w+");
	fwrite(h_results,size,1,outputImage);
	fclose(outputImage);

	hipEventElapsedTime(&time, start, stop);
	printf ("Time for the kernel: %f ms\n", time);

	return 0;
}